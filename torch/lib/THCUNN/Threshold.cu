#include "THCUNN.h"
#include "common.h"

struct ThresholdUpdateOutput
{
  const float threshold_;
  const float val_;

  ThresholdUpdateOutput(float threshold, float val)
    : threshold_(threshold)
    , val_(val)
  {}

  __device__ __forceinline__ void operator()(float *out, float *in)
  {
    float x = *in;
    *out = (x > threshold_) ? x : val_;
  }
};

// in-place variant
struct ThresholdUpdateOutputIP
{
  const float threshold_;
  const float val_;

  ThresholdUpdateOutputIP(float threshold, float val)
    : threshold_(threshold)
    , val_(val)
  {}

  __device__ __forceinline__ void operator()(float *x)
  {
    *x = (*x > threshold_) ? *x : val_;
  }
};

void THNN_CudaThreshold_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *output,
  double threshold, double val, bool inplace)
{
  THCUNN_assertSameGPU(state, 2, input, output);

  if (inplace)
  {
    THC_pointwiseApply1(state, input,
      ThresholdUpdateOutputIP(threshold, val)
    );
    THCudaTensor_set(state, output, input);
  }
  else
  {
    THCudaTensor_resizeAs(state, output, input);
    THC_pointwiseApply2(state, output, input,
      ThresholdUpdateOutput(threshold, val)
    );
  }

  THCudaCheck(hipGetLastError());
}

struct ThresholdUpdateGradInput
{
  const float threshold_;

  ThresholdUpdateGradInput(float threshold)
    : threshold_(threshold)
  {}

  __device__ __forceinline__ void operator()(
    float *gradInput, float *input, float *gradOutput) const
  {
    *gradInput = (*input > threshold_) ? *gradOutput : 0;
  }
};

struct ThresholdUpdateGradInputIP
{
  const float threshold_;

  ThresholdUpdateGradInputIP(float threshold)
    : threshold_(threshold)
  {}

  __device__ __forceinline__ void operator()(
    float *gradOutput, float *input) const
  {
    *gradOutput = (*input > threshold_) ? *gradOutput : 0;
  }
};

void THNN_CudaThreshold_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *gradOutput,
  THCudaTensor *gradInput, double threshold, double val, bool inplace)
{
  THCUNN_assertSameGPU(state, 3, input, gradInput, gradOutput);

  if (inplace)
  {
    THC_pointwiseApply2(state, gradOutput, input,
      ThresholdUpdateGradInputIP(threshold)
    );
    THCudaTensor_set(state, gradInput, gradOutput);
  }
  else
  {
    THCudaTensor_resizeAs(state, gradInput, input);
    THC_pointwiseApply3(state, gradInput, input, gradOutput,
       ThresholdUpdateGradInput(threshold)
    );
  }

  THCudaCheck(hipGetLastError());
}
