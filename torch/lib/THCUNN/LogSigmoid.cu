#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"

struct logSigmoid_updateOutput_functor
{
  __device__ void operator()(float *output, const float *input) const
  {
    float z = exp(-*input);
    *output = -log(1. + z);
  }
};

void THNN_CudaLogSigmoid_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *output, THCudaTensor *buffer)
{
  THCUNN_assertSameGPU(state, 2, input, output);
  THCudaTensor_resizeAs(state, output, input);
  THC_pointwiseApply2(state, output, input, logSigmoid_updateOutput_functor());
}

struct logSigmoid_updateGradInput_functor
{
  __device__ void operator()(float *gradInput, const float *input, const float *gradOutput) const
  {
    float z = exp(-*input);
    *gradInput = *gradOutput * z / (1. + z);
  }
};

void THNN_CudaLogSigmoid_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *gradOutput,
  THCudaTensor *gradInput , THCudaTensor *buffer)
{
  THCUNN_assertSameGPU(state, 3, input, gradOutput, gradInput);
  THCudaTensor_resizeAs(state, gradInput, input);
  THC_pointwiseApply3(state, gradInput, input, gradOutput, logSigmoid_updateGradInput_functor());
}
