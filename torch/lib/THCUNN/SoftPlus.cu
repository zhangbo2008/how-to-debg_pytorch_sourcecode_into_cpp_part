#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"

struct softPlusupdateOutput_functor
{
  const float threshold;
  const float beta;

  softPlusupdateOutput_functor(float threshold_, float beta_)
    : threshold(threshold_)
    , beta(beta_)
  {}

  __device__ void operator()(float *output, const float *input) const
  {
    float betain = beta * (*input);
    *output = ((betain) > threshold) ? *input : (1/beta) * log1p(exp(betain));
  }
};

void THNN_CudaSoftPlus_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *output, float beta, float threshold)
{
  THCUNN_assertSameGPU(state, 2, input, output);
  THCudaTensor_resizeAs(state, output, input);
  THC_pointwiseApply2(state, output, input, softPlusupdateOutput_functor(threshold, beta));
}

struct softPlusupdateGradInput_functor
{
  const float threshold;
  const float beta;

  softPlusupdateGradInput_functor(float threshold_, float beta_)
    : threshold(threshold_)
    , beta(beta_)
  {}

  __device__ void operator()(float *gradInput, const float *output, const float *gradOutput) const
  {
    float betaout = beta * (*output);
    float exp_bo = exp(betaout);
    *gradInput = ((betaout) > threshold) ? *gradOutput : *gradOutput * (exp_bo - 1) / exp_bo;
  }
};

void THNN_CudaSoftPlus_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *gradOutput, THCudaTensor *gradInput,
  THCudaTensor *output, float beta, float threshold)
{
  THCUNN_assertSameGPU(state, 4, input, output, gradOutput, gradInput);
  THCudaTensor_resizeAs(state, gradInput, output);
  THC_pointwiseApply3(state, gradInput, output, gradOutput, softPlusupdateGradInput_functor(threshold, beta));
}
