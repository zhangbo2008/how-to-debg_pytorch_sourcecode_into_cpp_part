#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"

struct sigmoidupdateOutput_functor
{
  __device__ void operator()(float *output, const float *input) const
  {
    *output = 1./(1.+ exp(-*input));
  }
};

void THNN_CudaSigmoid_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *output)
{
  THCUNN_assertSameGPU(state, 2, input, output);
  THCudaTensor_resizeAs(state, output, input);
  THC_pointwiseApply2(state, output, input, sigmoidupdateOutput_functor());
}

struct sigmoidupdateGradInput_functor
{
  __device__ void operator()(float *gradInput, const float *output, const float *gradOutput) const
  {
    *gradInput = *gradOutput * (1.-*output) * (*output);
  }
};

void THNN_CudaSigmoid_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *gradOutput, THCudaTensor *gradInput, THCudaTensor *output)
{
  THCUNN_assertSameGPU(state, 3, output, gradOutput, gradInput);
  THCudaTensor_resizeAs(state, gradInput, output);
  THC_pointwiseApply3(state, gradInput, output, gradOutput, sigmoidupdateGradInput_functor());
}
