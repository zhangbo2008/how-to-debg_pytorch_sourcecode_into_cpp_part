#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"
#include "THCReduceApplyUtils.cuh"

#include <thrust/functional.h>

#define MULTILABELMARGIN_THREADS 1024

__global__ void cunn_MultiLabelMarginCriterion_updateOutput_kernel(float *output,
                                                                   float *input,
                                                                   float *target,
                                                                   float *istarget,
                                                                   int nframe,
                                                                   int dim,
                                                                   int sizeaverage)
{
  // Temporary sums (for mapreduce)
  __shared__ float sums[MULTILABELMARGIN_THREADS];

  // vectors:
  int k = blockIdx.x;
  float *input_k = input + k*dim;
  float *target_k = target + k*dim;
  float *output_k = output + k;
  float *istarget_k = istarget + k*dim;

  // zero istarget
  for (int d = threadIdx.x; d < dim; d += blockDim.x) {
    istarget_k[d] = 0;
  }
  __syncthreads();

  // mark targets in istarget
  if (threadIdx.x == 0) {
    for (int dt = 0; dt < dim; dt++) {
      int target_idx = (int)target_k[dt] - TH_INDEX_BASE;
      if (target_idx < 0) break;
      istarget_k[target_idx] = 1;
    }
  }
  __syncthreads();

  // iterate over targets
  float sum = 0;
  for (int dt = 0; dt < dim; dt++) {
    // next target:
    int target_idx = (int)target_k[dt] - TH_INDEX_BASE;
    if (target_idx < 0) break;

    // current value for target
    float input_target_k = input_k[target_idx];

    // compare to all inputs (multithreaded):
    for (int d = threadIdx.x; d < dim; d += blockDim.x) {
      // contribute to loss only if not a target
      if (!istarget_k[d]) {
        float z = 1 - input_target_k + input_k[d];
        if (z > 0)
          sum += z;
      }
    }
  }

  // reduce
  float totalSum = reduceBlock(sums, blockDim.x, sum, thrust::plus<float>(), 0.0f);
  if (threadIdx.x == 0) {
    if (sizeaverage) {
      *output_k = (totalSum / dim) / nframe;
    } else {
      *output_k = totalSum / dim;
    }
  }
}

__global__ void cunn_MultiLabelMarginCriterion_updateGradInput_kernel(float *gradInput,
                                                                      float *input,
                                                                      float *target,
                                                                      float *istarget,
                                                                      int nframe,
                                                                      int dim,
                                                                      int sizeaverage)
{
  // Temporary sums (for mapreduce)
  __shared__ float sums[MULTILABELMARGIN_THREADS];

  // vectors:
  int k = blockIdx.x;
  float *input_k = input + k*dim;
  float *gradInput_k = gradInput + k*dim;
  float *target_k = target + k*dim;
  float *istarget_k = istarget + k*dim;

  // gain:
  float g = ( sizeaverage ? 1./((float)(nframe*dim)) : 1./((float)dim) );

  // zero gradients:
  for (int d = threadIdx.x; d < dim; d += blockDim.x) {
    gradInput_k[d] = 0;
  }
  __syncthreads();

  // iterate over targets
  for (int dt = 0; dt < dim; dt++) {
    // next target:
    int target_idx = (int)target_k[dt] - TH_INDEX_BASE;
    if (target_idx < 0) break;

    // current value for target
    float input_target_k = input_k[target_idx];

    // compare to all inputs (multithreaded):
    float sum = 0;
    for (int d = threadIdx.x; d < dim; d += blockDim.x) {
      // contribute to loss only if not a target
      if (!istarget_k[d]) {
        float z = 1 - input_target_k + input_k[d];
        if (z > 0) {
          sum -= g;
          gradInput_k[d] += g;
        }
      }
    }
    __syncthreads();

    // reduce sum
    float totalSum = reduceBlock(sums, blockDim.x, sum, thrust::plus<float>(), 0.0f);
    if (threadIdx.x == 0) {
      gradInput_k[target_idx] += totalSum;
    }
    __syncthreads();
  }
}

void THNN_CudaMultiLabelMarginCriterion_updateOutput(
          THCState *state,
          THCudaTensor *input,
          THCudaTensor *target,
          THCudaTensor *output,
          THCudaTensor *istarget,
          bool sizeaverage)
{
  input = THCudaTensor_newContiguous(state, input);
  target = THCudaTensor_newContiguous(state, target);
  istarget = THCudaTensor_newContiguous(state, istarget);
  THCudaTensor_resizeAs(state, istarget, input);

  if(input->nDimension == 1)
  {
    THCudaTensor_resize1d(state, output, 1);

    dim3 blocks(1);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateOutput_kernel<<<blocks,threads>>>(
        THCudaTensor_data(state, output),
        THCudaTensor_data(state, input),
        THCudaTensor_data(state, target),
        THCudaTensor_data(state, istarget),
        1, input->size[0],
        sizeaverage
        );
    THCudaCheck(hipGetLastError());
  }
  else if(input->nDimension == 2)
  {
    THCudaTensor *output_tmp = THCudaTensor_newWithSize1d(state, input->size[0]);

    dim3 blocks(input->size[0]);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateOutput_kernel<<<blocks,threads>>>(
        THCudaTensor_data(state, output_tmp),
        THCudaTensor_data(state, input),
        THCudaTensor_data(state, target),
        THCudaTensor_data(state, istarget),
        input->size[0], input->size[1],
        sizeaverage
        );
    THCudaCheck(hipGetLastError());
    THCudaTensor_resize1d(state, output, 1);
    THCudaTensor_set1d(state, output, 0, THCudaTensor_sumall(state, output_tmp));
    THCudaTensor_free(state, output_tmp);
  }
  else
    THError("vector or matrix expected");

  THCudaTensor_free(state, input);
  THCudaTensor_free(state, target);
  THCudaTensor_free(state, istarget);
}

void THNN_CudaMultiLabelMarginCriterion_updateGradInput(
          THCState *state,
          THCudaTensor *input,
          THCudaTensor *target,
          THCudaTensor *gradInput,
          THCudaTensor *istarget,
          bool sizeaverage)
{
  input = THCudaTensor_newContiguous(state, input);
  target = THCudaTensor_newContiguous(state, target);
  istarget = THCudaTensor_newContiguous(state, istarget);
  THCudaTensor_resizeAs(state, gradInput, input);

  if(gradInput->nDimension == 1)
  {
    dim3 blocks(1);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateGradInput_kernel<<<blocks,threads>>>(THCudaTensor_data(state, gradInput),
        THCudaTensor_data(state, input),
        THCudaTensor_data(state, target),
        THCudaTensor_data(state, istarget),
        1, gradInput->size[0],
        sizeaverage);

  }
  else if(gradInput->nDimension == 2)
  {
    dim3 blocks(gradInput->size[0]);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateGradInput_kernel<<<blocks,threads>>>(THCudaTensor_data(state, gradInput),
        THCudaTensor_data(state, input),
        THCudaTensor_data(state, target),
        THCudaTensor_data(state, istarget),
        gradInput->size[0], gradInput->size[1],
        sizeaverage);
  }
  else
    THError("vector or matrix expected");

  THCudaCheck(hipGetLastError());

  THCudaTensor_free(state, input);
  THCudaTensor_free(state, target);
  THCudaTensor_free(state, istarget);
}

#undef MULTILABELMARGIN_THREADS
