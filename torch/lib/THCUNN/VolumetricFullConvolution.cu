#include "THCUNN.h"
#include "common.h"
#include "vol2col.h"


void THNN_CudaVolumetricFullConvolution_updateOutput(
    THCState *state,
    THCudaTensor *input,
    THCudaTensor *output,
    THCudaTensor *weight,
    THCudaTensor *bias,
    THCudaTensor *finput,
    THCudaTensor *fgradInput,
    int dT, int dW, int dH,
    int padT, int padW, int padH,
    int adjT, int adjW, int adjH)
{

  THCudaTensor *columns = finput;
  THCudaTensor *ones    = fgradInput;

  int nInputPlane = THCudaTensor_size(state, weight, 0);
  int nOutputPlane = THCudaTensor_size(state, weight, 1);
  const int kT           = (int)weight->size[2];
  const int kH           = (int)weight->size[3];
  const int kW           = (int)weight->size[4];

  THCUNN_assertSameGPU(state, 6, input, output, weight,
                                 bias, columns, ones);
  THArgCheck(input->nDimension == 4 || input->nDimension == 5, 2, "4D or 5D (batch mode) tensor is expected");

  int batch = 1;
  if (input->nDimension == 4) {
    THArgCheck(input->size[0] == nInputPlane, 2, "input channels and nInputPlane dont match");
    // Force batch
    batch = 0;
    THCudaTensor_resize5d(state, input, 1, input->size[0], input->size[1], input->size[2], input->size[3]);
  } else {
    THArgCheck(input->size[1] == nInputPlane, 2, "input channels and nInputPlane dont match");
  }

  long inputWidth   = input->size[4];
  long inputHeight  = input->size[3];
  long inputDepth  = input->size[2];
  long outputWidth  = (inputWidth - 1) * dW - 2*padW + kW + adjW;
  long outputHeight = (inputHeight - 1) * dH - 2*padH + kH + adjH;
  long outputDepth = (inputDepth - 1) * dT - 2*padT + kT + adjT;

  // Batch size + input planes
  long batchSize = input->size[0];

  // Resize output
  THCudaTensor_resize5d(state, output, batchSize, nOutputPlane, outputDepth, outputHeight, outputWidth);

  // Resize temporary columns
  THCudaTensor_resize2d(state, columns, nOutputPlane*kW*kH*kT, inputDepth*inputHeight*inputWidth);

  // Define a buffer of ones, for bias accumulation
  // Note: this buffer can be shared with other modules, it only ever gets increased,
  // and always contains ones.
  if (ones->nDimension != 3 || ones->size[0]*ones->size[1]*ones->size[2] < outputDepth*outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCudaTensor_resize3d(state, ones, outputDepth, outputHeight, outputWidth);
    THCudaTensor_fill(state, ones, 1);
  }

  // Helpers
  THCudaTensor *input_n = THCudaTensor_new(state);
  THCudaTensor *output_n = THCudaTensor_new(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCudaTensor_select(state, input_n, input, 0, elt);
    THCudaTensor_select(state, output_n, output, 0, elt);

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m = weight->size[1] * weight->size[2] * weight->size[3] * weight->size[4];
    long n = columns->size[1];
    long k = weight->size[0];

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    THCudaBlas_Sgemm(
        state,
        'n', 't',
        n, m, k,
        1,
        THCudaTensor_data(state, input_n), n,
        THCudaTensor_data(state, weight), m,
        0,
        THCudaTensor_data(state, columns), n
    );

    // Unpack columns back into input:
    col2vol(
      THCState_getCurrentStream(state),
      THCudaTensor_data(state, columns),
      nOutputPlane, outputDepth, outputHeight, outputWidth, kT, kH, kW, padT, padH, padW, dT, dH, dW,
      1,1,1,
      THCudaTensor_data(state, output_n)
    );

    // Do Bias after:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m_ = nOutputPlane;
    long n_ = outputDepth * outputHeight * outputWidth;
    long k_ = 1;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    THCudaBlas_Sgemm(
        state,
        't', 'n',
        n_, m_, k_,
        1,
        THCudaTensor_data(state, ones), k_,
        THCudaTensor_data(state, bias), k_,
        1,
        THCudaTensor_data(state, output_n), n_
    );

  }

  // Free
  THCudaTensor_free(state, input_n);
  THCudaTensor_free(state, output_n);

  // Resize output
  if (batch == 0) {
    THCudaTensor_resize4d(state, output, nOutputPlane, outputDepth, outputHeight, outputWidth);
    THCudaTensor_resize4d(state, input, nInputPlane, inputDepth, inputHeight, inputWidth);
  }
}

void THNN_CudaVolumetricFullConvolution_updateGradInput(
    THCState *state,
    THCudaTensor *input,
    THCudaTensor *gradOutput,
    THCudaTensor *gradInput,
    THCudaTensor *weight,
    THCudaTensor *finput,
    THCudaTensor *fgradInput,
    int dT, int dW, int dH,
    int padT, int padW, int padH,
    int adjT, int adjW, int adjH)
{
  THCudaTensor *gradColumns = finput;

  int nInputPlane = THCudaTensor_size(state, weight, 0);
  int nOutputPlane = THCudaTensor_size(state, weight, 1);
  const int kT           = (int)weight->size[2];
  const int kH           = (int)weight->size[3];
  const int kW           = (int)weight->size[4];

  THCUNN_assertSameGPU(state, 5, input, gradOutput, weight,
                                 gradColumns, gradInput);
  THArgCheck(input->nDimension == 4 || input->nDimension == 5, 2, "4D or 5D (batch mode) tensor is expected");

  int batch = 1;
  if (input->nDimension == 4) {
    // Force batch
    batch = 0;
    THCudaTensor_resize5d(state, input, 1, input->size[0], input->size[1], input->size[2], input->size[3]);
    THCudaTensor_resize5d(state, gradOutput, 1, gradOutput->size[0], gradOutput->size[1], gradOutput->size[2], gradOutput->size[3]);
  }

  long inputWidth   = input->size[4];
  long inputHeight  = input->size[3];
  long inputDepth   = input->size[2];
  long outputWidth  = (inputWidth - 1) * dW - 2*padW + kW + adjW;
  long outputHeight = (inputHeight - 1) * dH - 2*padH + kH + adjH;
  long outputDepth = (inputDepth - 1) * dT - 2*padT + kT + adjT;

  // Batch size + input planes
  long batchSize = input->size[0];

  // Resize output
  THCudaTensor_resize5d(state, gradInput, batchSize, nInputPlane, inputDepth, inputHeight, inputWidth);

  // Resize temporary columns
  THCudaTensor_resize2d(state, gradColumns, nOutputPlane*kW*kH*kT, inputDepth*inputHeight*inputWidth);

  // Helpers
  THCudaTensor *gradInput_n = THCudaTensor_new(state);
  THCudaTensor *gradOutput_n = THCudaTensor_new(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per sample:
    THCudaTensor_select(state, gradInput_n, gradInput, 0, elt);
    THCudaTensor_select(state, gradOutput_n, gradOutput, 0, elt);

    // Extract columns:
    vol2col(
      THCState_getCurrentStream(state),
      THCudaTensor_data(state, gradOutput_n),
      nOutputPlane, outputDepth, outputHeight, outputWidth, kT, kH, kW, padT, padH, padW, dT, dH, dW,
      1,1,1,
      THCudaTensor_data(state, gradColumns)
    );


    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m = weight->size[0];
    long n = gradColumns->size[1];
    long k = weight->size[1] * weight->size[2] * weight->size[3] * weight->size[4];

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    THCudaBlas_Sgemm(
        state,
        'n', 'n',
        n, m, k,
        1,
        THCudaTensor_data(state, gradColumns), n,
        THCudaTensor_data(state, weight), k,
        0,
        THCudaTensor_data(state, gradInput_n), n
    );
  }


  // Free
  THCudaTensor_free(state, gradInput_n);
  THCudaTensor_free(state, gradOutput_n);

  // Resize output
  if (batch == 0) {
    THCudaTensor_resize4d(state, gradOutput, nOutputPlane, outputDepth, outputHeight, outputWidth);
    THCudaTensor_resize4d(state, input, nInputPlane, inputDepth, inputHeight, inputWidth);
    THCudaTensor_resize4d(state, gradInput, nInputPlane, inputDepth, inputHeight, inputWidth);
  }
}


void THNN_CudaVolumetricFullConvolution_accGradParameters(
    THCState *state,
    THCudaTensor *input,
    THCudaTensor *gradOutput,
    THCudaTensor *gradWeight,
    THCudaTensor *gradBias,
    THCudaTensor *finput,
    THCudaTensor *fgradInput,
    int dT, int dW, int dH,
    int padT, int padW, int padH,
    int adjT, int adjW, int adjH,
    float scale)
{
  THCudaTensor *columns = finput;
  THCudaTensor *ones = fgradInput;

  int nInputPlane = THCudaTensor_size(state, gradWeight, 0);
  int nOutputPlane = THCudaTensor_size(state, gradWeight, 1);
  const int kT           = (int)gradWeight->size[2];
  const int kH           = (int)gradWeight->size[3];
  const int kW           = (int)gradWeight->size[4];

  THCUNN_assertSameGPU(state, 6, input, gradOutput, gradWeight,
                                 gradBias, columns, ones);
  THArgCheck(input->nDimension == 4 || input->nDimension == 5, 2, "4D or 5D (batch mode) tensor is expected");

  int batch = 1;
  if (input->nDimension == 4) {
    // Force batch
    batch = 0;
    THCudaTensor_resize5d(state, input, 1, input->size[0], input->size[1], input->size[2], input->size[3]);
    THCudaTensor_resize5d(state, gradOutput, 1, gradOutput->size[0], gradOutput->size[1], gradOutput->size[2], gradOutput->size[3]);
  }

  long inputWidth   = input->size[4];
  long inputHeight  = input->size[3];
  long inputDepth   = input->size[2];
  long outputWidth  = (inputWidth - 1) * dW - 2*padW + kW + adjW;
  long outputHeight = (inputHeight - 1) * dH - 2*padH + kH + adjH;
  long outputDepth  = (inputDepth - 1) * dT - 2*padT + kT + adjT;

  // Batch size + input planes
  long batchSize = input->size[0];

  // Define a buffer of ones, for bias accumulation
  if (ones->nDimension != 3 || ones->size[0]*ones->size[1]*ones->size[2] < outputDepth*outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCudaTensor_resize3d(state, ones, outputDepth, outputHeight, outputWidth);
    THCudaTensor_fill(state, ones, 1);
  }

  // Resize temporary columns
  THCudaTensor_resize2d(state, columns, nOutputPlane*kW*kH*kT, inputDepth*inputHeight*inputWidth);

  // Helpers
  THCudaTensor *input_n = THCudaTensor_new(state);
  THCudaTensor *gradOutput_n = THCudaTensor_new(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCudaTensor_select(state, input_n, input, 0, elt);
    THCudaTensor_select(state, gradOutput_n, gradOutput, 0, elt);

    // Extract columns:
    vol2col(
      THCState_getCurrentStream(state),
      THCudaTensor_data(state, gradOutput_n),
      nOutputPlane, outputDepth, outputHeight, outputWidth, kT, kH, kW, padT, padH, padW, dT, dH, dW,
      1,1,1,
      THCudaTensor_data(state, columns)
    );

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long n = columns->size[0];   // nOutputPlane * kt * kh * kw
    long m = input_n->size[0];   // nInputPlane
    long k = columns->size[1];   // inputHeight * inputWidth

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    THCudaBlas_Sgemm(
        state,
        't', 'n',
        n, m, k,
        scale,
        THCudaTensor_data(state, columns), k,
        THCudaTensor_data(state, input_n), k,
        1,
        THCudaTensor_data(state, gradWeight), n
    );

    // Do Bias:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m_ = nOutputPlane;
    long k_ = outputDepth * outputHeight * outputWidth;

    // Do GEMV (note: this is a bit confusing because gemv assumes column-major matrices)
    THCudaBlas_Sgemv(
        state,
        't',
        k_, m_,
        scale,
        THCudaTensor_data(state, gradOutput_n), k_,
        THCudaTensor_data(state, ones), 1,
        1,
        THCudaTensor_data(state, gradBias), 1
    );
  }

  // Free
  THCudaTensor_free(state, input_n);
  THCudaTensor_free(state, gradOutput_n);

  // Resize
  if (batch == 0) {
    THCudaTensor_resize4d(state, gradOutput, nOutputPlane, outputDepth, outputHeight, outputWidth);
    THCudaTensor_resize4d(state, input, nInputPlane, inputDepth, inputHeight, inputWidth);
  }
}
