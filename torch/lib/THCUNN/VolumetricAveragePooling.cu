#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"
#include "THCDeviceTensor.cuh"
#include "THCDeviceTensorUtils.cuh"
#include "THCDeviceUtils.cuh"

__global__ void cuda_VolumetricAveragePooling_updateOutput(
  THCDeviceTensor<float, 4> input, THCDeviceTensor<float, 4> output,
  int kT, int kH, int kW, int dT, int dH, int dW, float normFactor, int offsetZ)
{
  int oCol   = blockIdx.x * blockDim.x + threadIdx.x;
  int oRow   = blockIdx.y * blockDim.y + threadIdx.y;
  int oFrame = (blockIdx.z + offsetZ) % output.getSize(1); // output frame/time
  int slice  = (blockIdx.z + offsetZ) / output.getSize(1); // output slice/feature

  if (oRow < output.getSize(2) && oCol < output.getSize(3))
  {
    float sum = 0.0;

    int iColumn = oCol * dW;
    int iRow    = oRow    * dH;
    int iFrame  = oFrame  * dT;

    for (int frame = 0; frame < kT; ++frame)
    {
      if (iFrame + frame < input.getSize(1))
      {
        for (int row = 0; row < kH; ++row)
        {
          if (iRow + row < input.getSize(2))
          {
            for (int column = 0; column < kW; ++column)
            {
              if (iColumn + column < input.getSize(3))
              {
                float val = input[slice][iFrame + frame][iRow + row][iColumn + column];
                sum += val;
              }
            }
          }
        }
      }
    }

    output[slice][oFrame][oRow][oCol] = sum * normFactor;
  }
}

// Inner-most loop size (kW) passed as template parameter for
// performance reasons.
//
template<int KERNEL_WIDTH>
__global__ void cuda_VolumetricAveragePooling_updateOutput(
  THCDeviceTensor<float, 4> input, THCDeviceTensor<float, 4> output,
  int kT, int kH, int dT, int dH, int dW, float normFactor, int offsetZ)
{
  int oCol   = blockIdx.x * blockDim.x + threadIdx.x;
  int oRow   = blockIdx.y * blockDim.y + threadIdx.y;
  int oFrame = (blockIdx.z + offsetZ) % output.getSize(1); // output frame/time
  int slice  = (blockIdx.z + offsetZ) / output.getSize(1); // output slice/feature

  if (oRow < output.getSize(2) && oCol < output.getSize(3))
  {
    float sum = 0.0;

    int iColumn = oCol * dW;
    int iRow    = oRow    * dH;
    int iFrame  = oFrame  * dT;

    for (int frame = 0; frame < kT; ++frame)
    {
      if (iFrame + frame < input.getSize(1))
      {
        for (int row = 0; row < kH; ++row)
        {
          if (iRow + row < input.getSize(2))
          {
            for (int column = 0; column < KERNEL_WIDTH; ++column)
            {
              if (iColumn + column < input.getSize(3))
              {
                float val = input[slice][iFrame + frame][iRow + row][iColumn + column];
                sum += val;
              }
            }
          }
        }
      }
    }

    output[slice][oFrame][oRow][oCol] = sum * normFactor;
  }
}

#define LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(KW) case KW:                  \
  cuda_VolumetricAveragePooling_updateOutput<KW><<<grid, block>>>(      \
    cudaInput, cudaOutput, kT, kH, dT, dH, dW, normFactor, offsetZ); \
  break


void THNN_CudaVolumetricAveragePooling_updateOutput(
  THCState *state, THCudaTensor *input, THCudaTensor *output,
  int kT, int kW, int kH,
  int dT, int dW, int dH)
{
  int batchSize;
  int inputSlices;
  int inputTime;
  int inputHeight;
  int inputWidth;

  if (THCudaTensor_nDimension(state, input) == 4)
  {
    THArgCheck(
      THCudaTensor_size(state, input, 1) >= kT &&
      THCudaTensor_size(state, input, 2) >= kH &&
      THCudaTensor_size(state, input, 3) >= kW, 2,
      "input image smaller than kernel size"
    );

    /* sizes */
    batchSize   = 1;
    inputSlices = THCudaTensor_size(state, input, 0);
    inputTime   = THCudaTensor_size(state, input, 1);
    inputHeight = THCudaTensor_size(state, input, 2);
    inputWidth  = THCudaTensor_size(state, input, 3);
  }
  else if (THCudaTensor_nDimension(state, input) == 5)
  {
    THArgCheck(
      THCudaTensor_size(state, input, 2) >= kT &&
      THCudaTensor_size(state, input, 3) >= kH &&
      THCudaTensor_size(state, input, 4) >= kW, 2,
      "input image smaller than kernel size"

    );
    /* sizes */
    batchSize   = THCudaTensor_size(state, input, 0);
    inputSlices = THCudaTensor_size(state, input, 1);
    inputTime   = THCudaTensor_size(state, input, 2);
    inputHeight = THCudaTensor_size(state, input, 3);
    inputWidth  = THCudaTensor_size(state, input, 4);
  }
  else
  {
    THArgCheck(false, 2, "4D or 5D tensor expected");
  }

  int outputTime   = (inputTime   - kT) / dT + 1;
  int outputHeight = (inputHeight - kH) / dH + 1;
  int outputWidth  = (inputWidth  - kW) / dW + 1;

  if (input->nDimension == 4) /* 4D */
  {
    /* resize output */
    THCudaTensor_resize4d(state, output, inputSlices,
                          outputTime, outputHeight, outputWidth);
  }
  else /* 5D */
  {
    THCudaTensor_resize5d(state, output, batchSize, inputSlices,
                          outputTime, outputHeight, outputWidth);
  }

  input = THCudaTensor_newContiguous(state, input);

  // Collapse batch and feature dimensions
  THCDeviceTensor<float, 4> cudaInput;
  THCDeviceTensor<float, 4> cudaOutput;
  if (THCudaTensor_nDimension(state, input) == 4)
  {
    cudaInput  = toDeviceTensor<float, 4>(state, input);
    cudaOutput = toDeviceTensor<float, 4>(state, output);
  }
  else
  {
    cudaInput  = toDeviceTensor<float, 5>(state, input).downcastOuter<4>();
    cudaOutput = toDeviceTensor<float, 5>(state, output).downcastOuter<4>();
  }

  int totalZ = outputTime * inputSlices * batchSize;
  int offsetZ = 0;
  dim3 block(32, 8);
  while (totalZ > 0) {
    dim3 grid(THCCeilDiv(outputWidth, static_cast<int>(block.x)),
              THCCeilDiv(outputHeight, static_cast<int>(block.y)),
              totalZ > 65535 ? 65535 : totalZ);

    float normFactor = 1.0f / static_cast<float>(kT * kH * kW);
    switch (kW)
      {
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(1);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(2);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(3);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(4);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(5);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(6);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(7);
      default:
        cuda_VolumetricAveragePooling_updateOutput<<<grid, block>>>(
                                                                    cudaInput,
                                                                    cudaOutput,
                                                                    kT, kH, kW,
                                                                    dT, dH, dW,
                                                                    normFactor,
                                                                    offsetZ
                                                                    );
        break;
      }
    totalZ -= 65535;
    offsetZ += 65535;
    THCudaCheck(hipGetLastError());
  }
  THCudaTensor_free(state, input);
}

__global__ void cuda_VolumetricAveragePooling_updateGradInput_Stride1(
  THCDeviceTensor<float, 4> gradOutput,
  THCDeviceTensor<float, 4> gradInput,
  int kT, int kH, int kW, float normFactor, int offsetZ)
{
  int iCol   = blockIdx.x * blockDim.x + threadIdx.x;
  int iRow   = blockIdx.y * blockDim.y + threadIdx.y;
  int iFrame = (blockIdx.z + offsetZ) % gradInput.getSize(1); // input frame/time
  int slice  = (blockIdx.z + offsetZ) / gradInput.getSize(1); // input slice/feature

  // guard against over-tiled threads
  if (iRow < gradInput.getSize(2) && iCol < gradInput.getSize(3))
  {
    float sum = 0.0;
    float *gOut = &gradOutput[slice][max(0, iFrame - kT + 1)]
      [max(0, iRow - kH + 1)][max(0, iCol - kW + 1)];
    int frameOffset = 0;
    for (int oFrame  = max(0, iFrame - kT + 1);
         oFrame < min(iFrame + 1, gradOutput.getSize(1));
         ++oFrame)
    {
      int rowOffset = frameOffset;
      for (int oRow = max(0, iRow - kH + 1);
           oRow < min(iRow + 1, gradOutput.getSize(2));
           ++oRow)
      {
        int colOffset = rowOffset;
        for (int oCol = max(0, iCol - kW + 1);
             oCol < min(iCol + 1, gradOutput.getSize(3));
             ++oCol)
        {
          sum += gOut[colOffset];
          ++colOffset;
        }
        rowOffset += gradOutput.getSize(3);
      }
      frameOffset += gradOutput.getSize(2) * gradOutput.getSize(3);
    }
    gradInput[slice][iFrame][iRow][iCol] = sum * normFactor;
  }
}

__global__ void cuda_VolumetricAveragePooling_updateGradInput_atomicAdd(
  THCDeviceTensor<float, 4> gradOutput,
  THCDeviceTensor<float, 4> gradInput,
  int kT, int kH, int kW, int dT, int dH, int dW, int offsetZ)
{
  int oCol   = blockIdx.x * blockDim.x + threadIdx.x;
  int oRow   = blockIdx.y * blockDim.y + threadIdx.y;
  int oFrame = (blockIdx.z + offsetZ) % gradOutput.getSize(1); // gradOutput frame/time
  int slice  = (blockIdx.z + offsetZ) / gradOutput.getSize(1); // gradOutput slice/feature

  // guard against over-tiled threads
  if (oRow < gradOutput.getSize(2) && oCol < gradOutput.getSize(3))
  {
    float val = gradOutput[slice][oFrame][oRow][oCol] / (kT * kH * kW);
    for (int iFrame = oFrame * dT; iFrame < oFrame * dT + kT; ++iFrame)
    {
      for (int iRow = oRow * dH; iRow < oRow * dH + kH; ++iRow)
      {
        for (int iCol = oCol * dW; iCol < oCol * dW + kW; ++iCol)
        {
          atomicAdd(&gradInput[slice][iFrame][iRow][iCol], val);
        }
      }
    }
  }
}

__global__ void cuda_VolumetricAveragePooling_updateGradInput(
  THCDeviceTensor<float, 4> gradOutput,
  THCDeviceTensor<float, 4> gradInput,
  int kT, int kH, int kW,
  int dT, int dH, int dW, int offsetZ)
{
  int oCol   = blockIdx.x * blockDim.x + threadIdx.x;
  int oRow   = blockIdx.y * blockDim.y + threadIdx.y;
  int oFrame = (blockIdx.z + offsetZ) % gradOutput.getSize(1); // gradOutput frame/time
  int slice  = (blockIdx.z + offsetZ) / gradOutput.getSize(1); // gradOutput slice/feature

  // guard against over-tiled threads
  if (oRow < gradOutput.getSize(2) && oCol < gradOutput.getSize(3))
  {
    float val = gradOutput[slice][oFrame][oRow][oCol] / (kT * kH * kW);
    for (int iFrame = oFrame * dT; iFrame < oFrame * dT + kT; ++iFrame)
    {
      for (int iRow = oRow * dH; iRow < oRow * dH + kH; ++iRow)
      {
        for (int iCol = oCol * dW; iCol < oCol * dW + kW; ++iCol)
        {
          gradInput[slice][iFrame][iRow][iCol] = val;
        }
      }
    }
  }
}

void THNN_CudaVolumetricAveragePooling_updateGradInput(
  THCState *state,
  THCudaTensor *input,
  THCudaTensor *gradOutput,
  THCudaTensor *gradInput,
  int kT, int kW, int kH,
  int dT, int dW, int dH)
{
  bool kernelsOverlap = (dT < kT) || (dH < kH) || (dW < kW);

  // Resize and initialize result tensor.
  THCudaTensor_resizeAs(state, gradInput, input);
  THCudaTensor_zero(state, gradInput);

  int batchSize;
  int inputSlices;
  int inputTime;
  int inputHeight;
  int inputWidth;

  int outputTime;
  int outputHeight;
  int outputWidth;

  if (THCudaTensor_nDimension(state, input) == 4) /* 4D */
  {
    batchSize = 1;
    inputSlices  = THCudaTensor_size(state, input, 0);
    inputTime    = THCudaTensor_size(state, input, 1);
    inputHeight  = THCudaTensor_size(state, input, 2);
    inputWidth   = THCudaTensor_size(state, input, 3);

    outputTime   = THCudaTensor_size(state, gradOutput, 1);
    outputHeight = THCudaTensor_size(state, gradOutput, 2);
    outputWidth  = THCudaTensor_size(state, gradOutput, 3);
  }
  else
  {
    batchSize    = THCudaTensor_size(state, input, 0);
    inputSlices  = THCudaTensor_size(state, input, 1);
    inputTime    = THCudaTensor_size(state, input, 2);
    inputHeight  = THCudaTensor_size(state, input, 3);
    inputWidth   = THCudaTensor_size(state, input, 4);

    outputTime   = THCudaTensor_size(state, gradOutput, 2);
    outputHeight = THCudaTensor_size(state, gradOutput, 3);
    outputWidth  = THCudaTensor_size(state, gradOutput, 4);
  }

  gradOutput = THCudaTensor_newContiguous(state, gradOutput);

  // Collapse batch and feature dimensions
  THCDeviceTensor<float, 4> cudaGradInput;
  THCDeviceTensor<float, 4> cudaGradOutput;
  if (THCudaTensor_nDimension(state, input) == 4)
  {
    cudaGradInput  = toDeviceTensor<float, 4>(state, gradInput);
    cudaGradOutput = toDeviceTensor<float, 4>(state, gradOutput);
  }
  else
  {
    cudaGradInput =
      toDeviceTensor<float, 5>(state, gradInput).downcastOuter<4>();
    cudaGradOutput =
      toDeviceTensor<float, 5>(state, gradOutput).downcastOuter<4>();
  }

  dim3 block(32, 8);

  // Optimizing for stride 1 is probably only of limited value, but this
  // specialization yields 3x speedup over the atomicAdd implementation.
  if (dT == 1 && dH == 1 && dW == 1)
  {
    int totalZ = inputTime * inputSlices * batchSize;
    int offsetZ = 0;
    while (totalZ > 0) {
      dim3 grid(THCCeilDiv(inputWidth, static_cast<int>(block.x)),
                THCCeilDiv(inputHeight, static_cast<int>(block.y)),
                totalZ > 65535 ? 65535 : totalZ);
      cuda_VolumetricAveragePooling_updateGradInput_Stride1<<<grid, block>>>(
         cudaGradOutput, cudaGradInput, kT, kH, kW, 1.0f/(kT * kH * kW), offsetZ);
      THCudaCheck(hipGetLastError());
      totalZ -= 65535;
      offsetZ += 65535;
    }
  }
  else
  {
    int totalZ = outputTime * inputSlices * batchSize;
    int offsetZ = 0;
    while (totalZ > 0) {

      dim3 grid(THCCeilDiv(outputWidth, static_cast<int>(block.x)),
                THCCeilDiv(outputHeight, static_cast<int>(block.y)),
                totalZ > 65535 ? 65535 : totalZ);
      if (kernelsOverlap)
        {
          cuda_VolumetricAveragePooling_updateGradInput_atomicAdd<<<grid, block>>>(
            cudaGradOutput, cudaGradInput, kT, kH, kW, dT, dH, dW, offsetZ);
        }
      else
        {
          cuda_VolumetricAveragePooling_updateGradInput<<<grid, block>>>(
             cudaGradOutput, cudaGradInput, kT, kH, kW, dT, dH, dW, offsetZ);
        }
      THCudaCheck(hipGetLastError());
      totalZ -= 65535;
      offsetZ += 65535;
    }
  }

  THCudaTensor_free(state, gradOutput);
}
