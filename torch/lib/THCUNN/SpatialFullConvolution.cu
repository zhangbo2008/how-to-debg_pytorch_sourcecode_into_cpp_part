#include "THCUNN.h"
#include "im2col.h"


void THNN_CudaSpatialFullConvolution_updateOutput(
    THCState *state,
    THCudaTensor *input,
    THCudaTensor *output,
    THCudaTensor *weight,
    THCudaTensor *bias,
    THCudaTensor *columns,
    THCudaTensor *ones,
    int kW, int kH,
    int dW, int dH,
    int padW, int padH,
    int adjW, int adjH)
{

  int nInputPlane = THCudaTensor_size(state, weight, 0);
  int nOutputPlane = THCudaTensor_size(state, weight, 1);

  THCUNN_assertSameGPU(state, 6, input, output, weight,
                                 bias, columns, ones);

  THArgCheck(input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch mode) tensor is expected");

  int batch = 1;
  if (input->nDimension == 3) {
    THArgCheck(input->size[0] == nInputPlane, 2, "input channels and nInputPlane dont match");
    // Force batch
    batch = 0;
    THCudaTensor_resize4d(state, input, 1, input->size[0], input->size[1], input->size[2]);
  } else {
    THArgCheck(input->size[1] == nInputPlane, 2, "input channels and nInputPlane dont match");
  }

  long inputWidth   = input->size[3];
  long inputHeight  = input->size[2];
  long outputWidth  = (inputWidth - 1) * dW - 2*padW + kW + adjW;
  long outputHeight = (inputHeight - 1) * dH - 2*padH + kH + adjH;

  // Batch size + input planes
  long batchSize = input->size[0];

  // Resize output
  THCudaTensor_resize4d(state, output, batchSize, nOutputPlane, outputHeight, outputWidth);

  // Resize temporary columns
  THCudaTensor_resize2d(state, columns, nOutputPlane*kW*kH, inputHeight*inputWidth);

  // Define a buffer of ones, for bias accumulation
  // Note: this buffer can be shared with other modules, it only ever gets increased,
  // and always contains ones.
  if (ones->nDimension != 2 || ones->size[0]*ones->size[1] < outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCudaTensor_resize2d(state, ones, outputHeight, outputWidth);
    THCudaTensor_fill(state, ones, 1);
  }

  // Helpers
  THCudaTensor *input_n = THCudaTensor_new(state);
  THCudaTensor *output_n = THCudaTensor_new(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCudaTensor_select(state, input_n, input, 0, elt);
    THCudaTensor_select(state, output_n, output, 0, elt);

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m = weight->size[1] * weight->size[2] * weight->size[3];
    long n = columns->size[1];
    long k = weight->size[0];

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    THCudaBlas_Sgemm(
        state,
        'n', 't',
        n, m, k,
        1,
        THCudaTensor_data(state, input_n), n,
        THCudaTensor_data(state, weight), m,
        0,
        THCudaTensor_data(state, columns), n
    );

    // Unpack columns back into input:
    col2im(
      THCState_getCurrentStream(state),
      THCudaTensor_data(state, columns),
      nOutputPlane, outputHeight, outputWidth, kH, kW, padH, padW, dH, dW,
      1, 1, THCudaTensor_data(state, output_n)
    );

    // Do Bias after:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m_ = nOutputPlane;
    long n_ = outputHeight * outputWidth;
    long k_ = 1;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    if (bias) {
      THCudaBlas_Sgemm(
          state,
          't', 'n',
          n_, m_, k_,
          1,
          THCudaTensor_data(state, ones), k_,
          THCudaTensor_data(state, bias), k_,
          1,
          THCudaTensor_data(state, output_n), n_
      );
    }

  }

  // Free
  THCudaTensor_free(state, input_n);
  THCudaTensor_free(state, output_n);

  // Resize output
  if (batch == 0) {
    THCudaTensor_resize3d(state, output, nOutputPlane, outputHeight, outputWidth);
    THCudaTensor_resize3d(state, input, nInputPlane, inputHeight, inputWidth);
  }
}

void THNN_CudaSpatialFullConvolution_updateGradInput(
    THCState *state,
    THCudaTensor *input,
    THCudaTensor *gradOutput,
    THCudaTensor *gradInput,
    THCudaTensor *weight,
    THCudaTensor *gradColumns,
    int kW, int kH,
    int dW, int dH,
    int padW, int padH,
    int adjW, int adjH)
{
  int nInputPlane = THCudaTensor_size(state, weight, 0);
  int nOutputPlane = THCudaTensor_size(state, weight, 1);

  THCUNN_assertSameGPU(state, 5, input, gradOutput, weight,
                                 gradColumns, gradInput);
  THArgCheck(input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch mode) tensor is expected");

  int batch = 1;
  if (input->nDimension == 3) {
    // Force batch
    batch = 0;
    THCudaTensor_resize4d(state, input, 1, input->size[0], input->size[1], input->size[2]);
    THCudaTensor_resize4d(state, gradOutput, 1, gradOutput->size[0], gradOutput->size[1], gradOutput->size[2]);
  }

  long inputWidth   = input->size[3];
  long inputHeight  = input->size[2];
  long outputWidth  = (inputWidth - 1) * dW - 2*padW + kW + adjW;
  long outputHeight = (inputHeight - 1) * dH - 2*padH + kH + adjH;

  // Batch size + input planes
  long batchSize = input->size[0];

  // Resize output
  THCudaTensor_resize4d(state, gradInput, batchSize, nInputPlane, inputHeight, inputWidth);

  // Resize temporary columns
  THCudaTensor_resize2d(state, gradColumns, nOutputPlane*kW*kH, inputHeight*inputWidth);

  // Helpers
  THCudaTensor *gradInput_n = THCudaTensor_new(state);
  THCudaTensor *gradOutput_n = THCudaTensor_new(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per sample:
    THCudaTensor_select(state, gradInput_n, gradInput, 0, elt);
    THCudaTensor_select(state, gradOutput_n, gradOutput, 0, elt);

    // Extract columns:
    im2col(
      THCState_getCurrentStream(state),
      THCudaTensor_data(state, gradOutput_n),
      nOutputPlane, outputHeight, outputWidth, kH, kW, padH, padW, dH, dW,
      1, 1, THCudaTensor_data(state, gradColumns)
    );


    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m = weight->size[0];
    long n = gradColumns->size[1];
    long k = weight->size[1] * weight->size[2] * weight->size[3];

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    THCudaBlas_Sgemm(
        state,
        'n', 'n',
        n, m, k,
        1,
        THCudaTensor_data(state, gradColumns), n,
        THCudaTensor_data(state, weight), k,
        0,
        THCudaTensor_data(state, gradInput_n), n
    );
  }


  // Free
  THCudaTensor_free(state, gradInput_n);
  THCudaTensor_free(state, gradOutput_n);

  // Resize output
  if (batch == 0) {
    THCudaTensor_resize3d(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
    THCudaTensor_resize3d(state, input, nInputPlane, inputHeight, inputWidth);
    THCudaTensor_resize3d(state, gradInput, nInputPlane, inputHeight, inputWidth);
  }
}


void THNN_CudaSpatialFullConvolution_accGradParameters(
    THCState *state,
    THCudaTensor *input,
    THCudaTensor *gradOutput,
    THCudaTensor *gradWeight,
    THCudaTensor *gradBias,
    THCudaTensor *columns,
    THCudaTensor *ones,
    int kW, int kH,
    int dW, int dH,
    int padW, int padH,
    int adjW, int adjH,
    float scale)
{
  int nInputPlane = THCudaTensor_size(state, gradWeight, 0);
  int nOutputPlane = THCudaTensor_size(state, gradWeight, 1);

  THCUNN_assertSameGPU(state, 6, input, gradOutput, gradWeight,
                                 gradBias, columns, ones);

  THArgCheck(input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch mode) tensor is expected");

  int batch = 1;
  if (input->nDimension == 3) {
    // Force batch
    batch = 0;
    THCudaTensor_resize4d(state, input, 1, input->size[0], input->size[1], input->size[2]);
    THCudaTensor_resize4d(state, gradOutput, 1, gradOutput->size[0], gradOutput->size[1], gradOutput->size[2]);
  }

  long inputWidth   = input->size[3];
  long inputHeight  = input->size[2];
  long outputWidth  = (inputWidth - 1) * dW - 2*padW + kW + adjW;
  long outputHeight = (inputHeight - 1) * dH - 2*padH + kH + adjH;

  // Batch size + input planes
  long batchSize = input->size[0];

  // Define a buffer of ones, for bias accumulation
  if (ones->nDimension != 2 || ones->size[0]*ones->size[1] < outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCudaTensor_resize2d(state, ones, outputHeight, outputWidth);
    THCudaTensor_fill(state, ones, 1);
  }

  // Resize temporary columns
  THCudaTensor_resize2d(state, columns, nOutputPlane*kW*kH, inputHeight*inputWidth);

  // Helpers
  THCudaTensor *input_n = THCudaTensor_new(state);
  THCudaTensor *gradOutput_n = THCudaTensor_new(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCudaTensor_select(state, input_n, input, 0, elt);
    THCudaTensor_select(state, gradOutput_n, gradOutput, 0, elt);

    // Extract columns:
    im2col(
      THCState_getCurrentStream(state),
      THCudaTensor_data(state, gradOutput_n),
      nOutputPlane, outputHeight, outputWidth, kH, kW, padH, padW, dH, dW,
      1, 1, THCudaTensor_data(state, columns)
    );

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long n = columns->size[0];   // nOutputPlane * kh * kw
    long m = input_n->size[0];   // nInputPlane
    long k = columns->size[1];   // inputHeight * inputWidth

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    THCudaBlas_Sgemm(
        state,
        't', 'n',
        n, m, k,
        scale,
        THCudaTensor_data(state, columns), k,
        THCudaTensor_data(state, input_n), k,
        1,
        THCudaTensor_data(state, gradWeight), n
    );

    // Do Bias:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m_ = nOutputPlane;
    long k_ = outputHeight * outputWidth;

    // Do GEMV (note: this is a bit confusing because gemv assumes column-major matrices)
    if (gradBias) {
      THCudaBlas_Sgemv(
          state,
          't',
          k_, m_,
          scale,
          THCudaTensor_data(state, gradOutput_n), k_,
          THCudaTensor_data(state, ones), 1,
          1,
          THCudaTensor_data(state, gradBias), 1
      );
    }
  }

  // Free
  THCudaTensor_free(state, input_n);
  THCudaTensor_free(state, gradOutput_n);

  // Resize
  if (batch == 0) {
    THCudaTensor_resize3d(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
    THCudaTensor_resize3d(state, input, nInputPlane, inputHeight, inputWidth);
  }
}
