#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"

template <typename Dtype>
__global__ void MaxUnpoolForward(const int nthreads, const Dtype* bottom_data, const Dtype* bottom_mask,
    const int num, const int channels, const int iheight, const int iwidth, const int oheight, const int owidth, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) { //index here indices the input pixels
    int c = (index / iwidth / iheight) % channels;
    int n = index / iwidth / iheight / channels;
    top_data += (n*channels + c)*oheight*owidth;
    int maxind = bottom_mask[index] - TH_INDEX_BASE;

    top_data[maxind] = bottom_data[index];
  }
}

template <typename Dtype>
__global__ void MaxUnpoolBackward(const int nthreads, const Dtype* top_diff, const Dtype* bottom_mask,
    const int num, const int channels, const int iheight, const int iwidth, const int oheight, const int owidth, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int c = (index / iwidth / iheight) % channels;
    int n = index / iwidth / iheight / channels;
    top_diff += (n*channels + c)*oheight*owidth;
    int maxind = bottom_mask[index] - TH_INDEX_BASE;

    bottom_diff[index] = top_diff[maxind];
  }
}

void THNN_CudaSpatialMaxUnpooling_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *output, THCudaTensor *indices, int owidth, int oheight)
{
  THCUNN_assertSameGPU(state, 3, input, output, indices);
  THArgCheck(input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch) tensor expected");

  long nInputCols, nInputRows, nInputPlane, batchSize;

  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }

  input = THCudaTensor_newContiguous(state, input);
  indices = THCudaTensor_newContiguous(state, indices);
  THCudaTensor_resize4d(state, output, batchSize, nInputPlane, oheight, owidth);
  THCudaTensor_zero(state, output);

  int count = THCudaTensor_nElement(state, input);

  MaxUnpoolForward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, THCudaTensor_data(state, input), THCudaTensor_data(state, indices),
      batchSize, nInputPlane, nInputRows, nInputCols, oheight, owidth, THCudaTensor_data(state, output));
  THCudaCheck(hipGetLastError());

  if(input->nDimension == 3)
    THCudaTensor_resize3d(state, output, nInputPlane, oheight, owidth);

  THCudaTensor_free(state, input);

}

void THNN_CudaSpatialMaxUnpooling_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *gradOutput, THCudaTensor *gradInput, THCudaTensor *indices, int owidth, int oheight)
{
  THCUNN_assertSameGPU(state, 4, input, gradOutput, indices, gradInput);

  long nInputCols, nInputRows, nInputPlane, batchSize;

  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }

  input = THCudaTensor_newContiguous(state, input);
  indices = THCudaTensor_newContiguous(state, indices);
  gradOutput = THCudaTensor_newContiguous(state, gradOutput);
  THCudaTensor_resizeAs(state, gradInput, input);

  int count = THCudaTensor_nElement(state, input);

  MaxUnpoolBackward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, THCudaTensor_data(state, gradOutput), THCudaTensor_data(state, indices),
      batchSize, nInputPlane, nInputRows, nInputCols, oheight, owidth, THCudaTensor_data(state, gradInput));
  THCudaCheck(hipGetLastError());

  // clean
  THCudaTensor_free(state, input);
  THCudaTensor_free(state, gradOutput);
}
