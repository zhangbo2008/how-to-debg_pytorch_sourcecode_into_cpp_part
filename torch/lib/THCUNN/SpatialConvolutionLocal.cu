#include "THCUNN.h"
#include "common.h"
#include "im2col.h"

void THNN_CudaSpatialConvolutionLocal_updateOutput(
    THCState *state,
    THCudaTensor *input,
    THCudaTensor *output,
    THCudaTensor *weight,
    THCudaTensor *bias,
    THCudaTensor *finput,
    THCudaTensor *fgradInput,
    int kW, int kH,
    int dW, int dH,
    int padW, int padH,
    long inputWidth, long inputHeight,
    long outputWidth, long outputHeight)
{
  THCUNN_assertSameGPU(state, 5, input, output, weight,
                                 bias, finput);

  long nInputPlane = THCudaTensor_size(state,weight,2)/(kW*kH);
  long nOutputPlane = THCudaTensor_size(state,weight,1);

  int batch = 1;
  if (input->nDimension == 3) {
    // Force batch
    batch = 0;
    THCudaTensor_resize4d(state, input, 1, nInputPlane, inputHeight, inputWidth);
  }

  // Batch size + input planes
  long batchSize = input->size[0];

  // Resize output
  THCudaTensor_resize4d(state, output, batchSize, nOutputPlane, outputHeight, outputWidth);

  // Augment the input
  THCudaTensor_resize3d(state, finput, batchSize, nInputPlane*kW*kH, outputHeight*outputWidth);

  // Helpers
  THCudaTensor *input_n = THCudaTensor_new(state);
  THCudaTensor *finput_n = THCudaTensor_new(state);
  THCudaTensor *output_n = THCudaTensor_new(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    THCudaTensor *finput3d, *output3d;
    THCudaTensor *wslice = THCudaTensor_new(state);
    THCudaTensor *islice = THCudaTensor_new(state);
    THCudaTensor *oslice = THCudaTensor_new(state);

    // Matrix mulitply per output:
    THCudaTensor_select(state, input_n, input, 0, elt);
    THCudaTensor_select(state, finput_n, finput, 0, elt);
    THCudaTensor_select(state, output_n, output, 0, elt);

    // Extract columns:
    im2col(
      THCState_getCurrentStream(state),
      THCudaTensor_data(state, input_n),
      nInputPlane, inputHeight, inputWidth, kH, kW, padH, padW, dH, dW,
      1, 1, THCudaTensor_data(state, finput_n)
    );

    output3d = THCudaTensor_newWithStorage3d(state, output_n->storage, output_n->storageOffset,
                                             outputHeight*outputWidth, 1,
                                             nOutputPlane, outputHeight*outputWidth,
                                             1, nOutputPlane*outputHeight*outputWidth);

    finput3d = THCudaTensor_newWithStorage3d(state, finput_n->storage, finput_n->storageOffset,
                                             outputHeight*outputWidth, 1,
                                             kW*kH*nInputPlane, outputHeight*outputWidth,
                                             1, kW*kH*nInputPlane*outputHeight*outputWidth);

    THCudaTensor_copy(state, output_n, bias);

    for (int i = 0; i < outputHeight; i++) {
      for(int j = 0; j < outputWidth; j++) {
        int sliceidx = i * outputWidth + j;
        THCudaTensor_select(state, wslice, weight, 0, sliceidx);
        THCudaTensor_select(state, islice, finput3d, 0, sliceidx);
        THCudaTensor_select(state, oslice, output3d, 0, sliceidx);
        THCudaTensor_addmm(state, oslice, 1.0, oslice, 1.0, wslice, islice);
      }
    }


    // weight:    oH*oW x nOutputPlane x nInputPlane*kH*kW
    // finput3d:  oH*oW x nInputPlane*kH*kW x 1
    // THCudaTensor_baddbmm(state, output3d, 1.0, output3d, 1.0, weight, finput3d);
    // output3d:  oH*oW x nOutputPlane x 1

    THCudaTensor_free(state, output3d);
    THCudaTensor_free(state, finput3d);
    THCudaTensor_free(state, wslice);
    THCudaTensor_free(state, islice);
    THCudaTensor_free(state, oslice);
  }

  // Free
  THCudaTensor_free(state, input_n);
  THCudaTensor_free(state, finput_n);
  THCudaTensor_free(state, output_n);

  // Resize output
  if (batch == 0) {
    THCudaTensor_resize3d(state, output, nOutputPlane, outputHeight, outputWidth);
    THCudaTensor_resize3d(state, input, nInputPlane, inputHeight, inputWidth);
  }
}

void THNN_CudaSpatialConvolutionLocal_updateGradInput(
    THCState *state,
    THCudaTensor *input,
    THCudaTensor *gradOutput,
    THCudaTensor *gradInput,
    THCudaTensor *weight,
    THCudaTensor *finput,
    THCudaTensor *fgradInput,
    int kW, int kH,
    int dW, int dH,
    int padW, int padH,
    long inputWidth, long inputHeight,
    long outputWidth, long outputHeight)
{
  THCUNN_assertSameGPU(state, 5, input, gradOutput, weight,
                                 fgradInput, gradInput);

  long nInputPlane = THCudaTensor_size(state,weight,2)/(kW*kH);
  long nOutputPlane = THCudaTensor_size(state,weight,1);

  int batch = 1;
  if (input->nDimension == 3) {
    // Force batch
    batch = 0;
    THCudaTensor_resize4d(state, input, 1, nInputPlane, inputHeight, inputWidth);
    THCudaTensor_resize4d(state, gradOutput, 1, nOutputPlane, outputHeight, outputWidth);
  }

  // Batch size + input planes
  long batchSize = input->size[0];

  // Resize output
  THCudaTensor_resize4d(state, gradInput, batchSize, nInputPlane, inputHeight, inputWidth);

  // Resize temporary columns
  THCudaTensor_resize3d(state, fgradInput, batchSize, nInputPlane*kW*kH, outputHeight*outputWidth);

  // Helpers
  THCudaTensor *gradInput_n = THCudaTensor_new(state);
  THCudaTensor *fgradInput_n = THCudaTensor_new(state);
  THCudaTensor *gradOutput_n = THCudaTensor_new(state);

  THCudaTensor_transpose(state, weight, weight, 1, 2);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    THCudaTensor *gradOutput3d, *fgradInput3d;
    THCudaTensor *wslice = THCudaTensor_new(state);
    THCudaTensor *gislice = THCudaTensor_new(state);
    THCudaTensor *goslice = THCudaTensor_new(state);

    // Matrix mulitply per sample:
    THCudaTensor_select(state, gradInput_n, gradInput, 0, elt);
    THCudaTensor_select(state, fgradInput_n, fgradInput, 0, elt);
    THCudaTensor_select(state, gradOutput_n, gradOutput, 0, elt);

    gradOutput3d = THCudaTensor_newWithStorage3d(state, gradOutput_n->storage, gradOutput_n->storageOffset,
                                               outputHeight*outputWidth, 1,
                                               nOutputPlane, outputHeight*outputWidth,
                                               1, nOutputPlane*outputHeight*outputWidth);
    fgradInput3d = THCudaTensor_newWithStorage3d(state, fgradInput_n->storage, fgradInput_n->storageOffset,
                                               outputHeight*outputWidth, 1,
                                               kW*kH*nInputPlane, outputHeight*outputWidth,
                                               1, kW*kH*nInputPlane*outputHeight*outputWidth);

    for (int i = 0; i < outputHeight; i++) {
      for(int j = 0; j < outputWidth; j++) {
        int sliceidx = i * outputWidth + j;
        THCudaTensor_select(state, wslice, weight, 0, sliceidx);
        THCudaTensor_select(state, gislice, fgradInput3d, 0, sliceidx);
        THCudaTensor_select(state, goslice, gradOutput3d, 0, sliceidx);
        THCudaTensor_addmm(state, gislice, 0.0, gislice, 1.0, wslice, goslice);
      }
    }

    // weight:        oH*oW x nInputPlane*kH*kW x nOutputPlane
    // gradOutput3d:  oH*oW x nOutputPlane x 1
    //THCudaTensor_baddbmm(state, fgradInput3d, 0.0, fgradInput3d, 1.0, weight, gradOutput3d);
    // fgradInput3d:  oH*oW x nInputPlane*kH*kW x 1

    // Unpack columns back into input:
    col2im(
      THCState_getCurrentStream(state),
      THCudaTensor_data(state, fgradInput_n),
      nInputPlane, inputHeight, inputWidth, kH, kW, padH, padW, dH, dW,
      1, 1, THCudaTensor_data(state, gradInput_n)
    );

    THCudaTensor_free(state, gradOutput3d);
    THCudaTensor_free(state, fgradInput3d);
    THCudaTensor_free(state, wslice);
    THCudaTensor_free(state, gislice);
    THCudaTensor_free(state, goslice);
  }

  // Free
  THCudaTensor_free(state, gradInput_n);
  THCudaTensor_free(state, fgradInput_n);
  THCudaTensor_free(state, gradOutput_n);

  // Resize output
  if (batch == 0) {
    THCudaTensor_resize3d(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
    THCudaTensor_resize3d(state, input, nInputPlane, inputHeight, inputWidth);
    THCudaTensor_resize3d(state, gradInput, nInputPlane, inputHeight, inputWidth);
  }

  THCudaTensor_transpose(state, weight, weight, 1, 2);
}

void THNN_CudaSpatialConvolutionLocal_accGradParameters(
    THCState *state,
    THCudaTensor *input,
    THCudaTensor *gradOutput,
    THCudaTensor *gradWeight,
    THCudaTensor *gradBias,
    THCudaTensor *finput,
    THCudaTensor *fgradInput,
    int kW, int kH,
    int dW, int dH,
    int padW, int padH,
    long inputWidth, long inputHeight,
    long outputWidth, long outputHeight,
    float scale)
{
  THCUNN_assertSameGPU(state, 5, input, gradOutput, gradWeight,
                                 gradBias, finput);

  long nInputPlane = THCudaTensor_size(state,gradWeight,2)/(kW*kH);
  long nOutputPlane = THCudaTensor_size(state,gradWeight,1);

  int batch = 1;
  if (input->nDimension == 3) {
    // Force batch
    batch = 0;
    THCudaTensor_resize4d(state, input, 1, nInputPlane, inputHeight, inputWidth);
    THCudaTensor_resize4d(state, gradOutput, 1, nOutputPlane, outputHeight, outputWidth);
  }

  // Batch size + input planes
  long batchSize = input->size[0];

  // Helpers
  THCudaTensor *input_n = THCudaTensor_new(state);
  THCudaTensor *finput_n = THCudaTensor_new(state);
  THCudaTensor *gradOutput_n = THCudaTensor_new(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    THCudaTensor *gradOutput3d, *finput3d;
    THCudaTensor *gwslice = THCudaTensor_new(state);
    THCudaTensor *islice = THCudaTensor_new(state);
    THCudaTensor *goslice = THCudaTensor_new(state);

    // Matrix mulitply per output:
    THCudaTensor_select(state, input_n, input, 0, elt);
    THCudaTensor_select(state, finput_n, finput, 0, elt);
    THCudaTensor_select(state, gradOutput_n, gradOutput, 0, elt);

    gradOutput3d = THCudaTensor_newWithStorage3d(state, gradOutput_n->storage, gradOutput_n->storageOffset,
                                                 outputHeight*outputWidth, 1,
                                                 nOutputPlane, outputHeight*outputWidth,
                                                 1, nOutputPlane*outputHeight*outputWidth);
    finput3d = THCudaTensor_newWithStorage3d(state, finput_n->storage, finput_n->storageOffset,
                                             outputHeight*outputWidth, 1,
                                             1, kW*kH*nInputPlane*outputHeight*outputWidth,
                                             kW*kH*nInputPlane, outputHeight*outputWidth);

    // Extract columns:
    im2col(
      THCState_getCurrentStream(state),
      THCudaTensor_data(state, input_n),
      nInputPlane, inputHeight, inputWidth, kH, kW, padH, padW, dH, dW,
      1, 1, THCudaTensor_data(state, finput_n)
    );

    for (int i = 0; i < outputHeight; i++) {
      for(int j = 0; j < outputWidth; j++) {
        int sliceidx = i * outputWidth + j;
        THCudaTensor_select(state, gwslice, gradWeight, 0, sliceidx);
        THCudaTensor_select(state, goslice, gradOutput3d, 0, sliceidx);
        THCudaTensor_select(state, islice, finput3d, 0, sliceidx);
        THCudaTensor_addmm(state, gwslice, 1.0, gwslice, scale, goslice, islice);
      }
    }
    // gradOutput3d:  oH*oW x nOutputPlane x 1
    // finput3d:      oH*oW x 1 x kW*kH*nInputPlane
    //THCudaTensor_baddbmm(state, gradWeight, 1.0, gradWeight, scale, gradOutput3d, finput3d);
    // gradWeight:    oH*oW x nOutputPlane x kW*kH*nInputPlane

    THCudaTensor_cadd(state, gradBias, gradBias, scale, gradOutput_n);

    THCudaTensor_free(state, gradOutput3d);
    THCudaTensor_free(state, finput3d);
    THCudaTensor_free(state, gwslice);
    THCudaTensor_free(state, goslice);
    THCudaTensor_free(state, islice);
  }

  // Free
  THCudaTensor_free(state, input_n);
  THCudaTensor_free(state, finput_n);
  THCudaTensor_free(state, gradOutput_n);

  // Resize
  if (batch == 0) {
    THCudaTensor_resize3d(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
    THCudaTensor_resize3d(state, input, nInputPlane, inputHeight, inputWidth);
  }
}
