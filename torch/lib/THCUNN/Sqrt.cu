#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"

struct sqrtupdateOutput_functor
{
  const float bias;

  sqrtupdateOutput_functor(float bias_)
    : bias(bias_)
  {}

  __device__ void operator()(float *output, const float *input) const
  {
    *output = sqrt(*input + bias);
  }
};

void THNN_CudaSqrt_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *output, float eps)
{
  THCUNN_assertSameGPU(state, 2, input, output);
  THCudaTensor_resizeAs(state, output, input);
  THC_pointwiseApply2(state, output, input, sqrtupdateOutput_functor(eps));
}

struct sqrtupdateGradInput_functor
{
  sqrtupdateGradInput_functor() {}

  __device__ void operator()(float *gradInput, const float *output, const float *gradOutput) const
  {
    *gradInput = (*output == 0.0f) ? 0.0f : ((0.5f * *gradOutput) / *output);
  }
};

void THNN_CudaSqrt_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *gradOutput, THCudaTensor *gradInput, THCudaTensor *output)
{
  THCUNN_assertSameGPU(state, 3, output, gradOutput, gradInput);
  THCudaTensor_resizeAs(state, gradInput, output);
  THC_pointwiseApply3(state, gradInput, output, gradOutput, sqrtupdateGradInput_functor());
}
