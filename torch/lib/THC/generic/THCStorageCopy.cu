
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCStorageCopy.cu"
#else

void THCStorage_(rawCopy)(THCState *state, THCStorage *self, real *src)
{
  THCudaCheck(cudaMemcpyAsync(self->data, src, self->size * sizeof(real), cudaMemcpyDeviceToDevice, THCState_getCurrentStream(state)));
}

void THCStorage_(copy)(THCState *state, THCStorage *self, THCStorage *src)
{
  THArgCheck(self->size == src->size, 2, "size does not match");
  THCudaCheck(cudaMemcpyAsync(self->data, src->data, self->size * sizeof(real), cudaMemcpyDeviceToDevice, THCState_getCurrentStream(state)));
}

void THCStorage_(copyCuda)(THCState *state, THCStorage *self, THCStorage *src)
{
  THArgCheck(self->size == src->size, 2, "size does not match");
  THCudaCheck(cudaMemcpyAsync(self->data, src->data, self->size * sizeof(real), cudaMemcpyDeviceToDevice, THCState_getCurrentStream(state)));
}

// conversions are mediated by the CPU
// yes, this is slow; feel free to write CUDA kernels for this
#ifndef THC_REAL_IS_HALF
#define THC_CUDA_STORAGE_IMPLEMENT_COPY(TYPEC,TYPECUDA)                            \
  void THCStorage_(copyCuda##TYPEC)(THCState *state, THCStorage *self, struct THCuda##TYPECUDA##Storage *src)  \
  {                                                                      \
    if(THCTypeIdx_(Real) == THCTypeIdx_(TYPEC)) {                        \
      THCStorage_(copy)(state, self, (THCStorage*) src);   /* cast just removes compiler warning */ \
    } else {                                                             \
      THArgCheck(self->size == src->size, 2, "size does not match");     \
      TH##TYPEC##Storage *buffer1 = TH##TYPEC##Storage_newWithSize(src->size); \
      THStorage *buffer2  = THStorage_(newWithSize)(src->size);          \
      TH##TYPEC##Storage_copyCuda(state, buffer1, src);                  \
      THStorage_(copy##TYPEC)(buffer2, buffer1);                         \
      THCStorage_(copyCPU)(state, self, buffer2);                        \
      TH##TYPEC##Storage_free(buffer1);                                  \
      THStorage_(free)(buffer2);                                         \
    }                                                                    \
  }
#else
#define THC_CUDA_STORAGE_IMPLEMENT_COPY(TYPEC,TYPECUDA)                            \
  void THCStorage_(copyCuda##TYPEC)(THCState *state, THCStorage *self, struct THCuda##TYPECUDA##Storage *src)  \
  {                                                                      \
    THArgCheck(self->size == src->size, 2, "size does not match");       \
    if(THCTypeIdx_(TYPEC) == THCTypeIdxFloat) {                          \
      THCFloat2Half(state, self->data, (float*) src->data, src->size);   /* cast removes compiler error */     \
    } else {                                                             \
      THCudaStorage *buffer = THCudaStorage_newWithSize(state, src->size); \
      THCudaStorage_copyCuda##TYPEC(state, buffer, src);                 \
      THCFloat2Half(state, self->data, buffer->data, buffer->size);      \
      THCudaStorage_free(state, buffer);                                 \
    }                                                                    \
  }
#endif

THC_CUDA_STORAGE_IMPLEMENT_COPY(Byte,Byte)
THC_CUDA_STORAGE_IMPLEMENT_COPY(Char,Char)
THC_CUDA_STORAGE_IMPLEMENT_COPY(Short,Short)
THC_CUDA_STORAGE_IMPLEMENT_COPY(Int,Int)
THC_CUDA_STORAGE_IMPLEMENT_COPY(Long,Long)
THC_CUDA_STORAGE_IMPLEMENT_COPY(Float,)  // i.e. float
THC_CUDA_STORAGE_IMPLEMENT_COPY(Double,Double)

#ifdef CUDA_HALF_TENSOR
#define FLOAT_COPY(TYPE) TH_CONCAT_3(TH, CReal, Storage_copyCudaFloat)
void THCStorage_(copyCudaHalf)(THCState *state, THCStorage *self, struct THCudaHalfStorage *src)
{
    if(THCTypeIdx_(Real) == THCTypeIdxHalf) {
      THCStorage_(copy)(state, self, (THCStorage*) src);   /* cast just removes compiler warning */
    } else {
        THArgCheck(self->size == src->size, 2, "size does not match");
        THCudaStorage *buffer = THCudaStorage_newWithSize(state, src->size);
        THCHalf2Float(state, buffer->data, src->data, src->size);
        FLOAT_COPY(Real)(state, self, buffer);
        THCudaStorage_free(state, buffer);
    }
}
#undef FLOAT_COPY
#endif // CUDA_HALF_TENSOR

#undef THC_CUDA_STORAGE_IMPLEMENT_COPY

#endif
