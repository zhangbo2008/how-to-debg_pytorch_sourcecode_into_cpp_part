
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMathPairwise.cu"
#else

THC_API void
THCTensor_(add)(THCState *state, THCTensor *self_, THCTensor *src_, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, self_, TensorAddConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2(state, self_, src_, TensorAddConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(sub)(THCState *state, THCTensor *self_, THCTensor *src_, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, self_, TensorSubConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2(state, self_, src_, TensorSubConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(mul)(THCState *state, THCTensor *self_, THCTensor *src_, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, self_, TensorMulConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2(state, self_, src_, TensorMulConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(div)(THCState* state, THCTensor *self_, THCTensor *src_, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 2, self_, src_));
  THArgCheck(value != ScalarConvert<int, real>::to(0), 3, "divide by zero");

  if (self_ == src_) {
    if (!THC_pointwiseApply1(state, self_, TensorDivConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2(state, self_, src_, TensorDivConstantOp<real>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

#endif
