#include "hip/hip_runtime.h"
#include "THC.h"
#include "THCReduceApplyUtils.cuh"
#include "THCTensorCopy.h"
#include "THCTensorMath.h"
#include "THCAsmUtils.cuh"
#include "THCScanUtils.cuh"
#include "THCTensorTypeUtils.cuh"
#include <algorithm> // for std::min

#if CUDA_VERSION >= 7000
#include <thrust/system/cuda/execution_policy.h>
#endif

// Converts a float to an integer representation with the same
// sorting; i.e., for floats f1, f2:
// if f1 < f2 then convert(f1) < convert(f2)
// We use this to enable radix selection of floating-point values.
// This also gives a relative order for NaNs, but that's ok, as they
// will all be adjacent
struct FloatToSortedInt {
  inline __host__ __device__ FloatToSortedInt() {}

  inline __device__ unsigned int convert(float v) const {
    unsigned int x = __float_as_int(v);
    unsigned int mask = (x & 0x80000000) ? 0xffffffff : 0x80000000;

    return (x ^ mask);
  }

  inline __device__ float deconvert(unsigned int v) const {
    unsigned int mask = (v & 0x80000000) ? 0x80000000 : 0xffffffff;

    return __int_as_float(v ^ mask);
  }
};

// This function counts the distribution of all input values in a
// slice we are selecting by radix digit at `radixDigitPos`, but only
// those that pass the filter `((v & desiredMask) == desired)`.
// This produces and broadcasts the seen counts for a single block only.
// `smem` must have at least `RadixSize` elements.
template <typename DataType, typename BitDataType,
          typename IndexType, typename CountType,
          typename RadixConverter, int RadixSize, int RadixBits>
__device__ void countRadixUsingMask(const RadixConverter& conv,
                                    CountType counts[RadixSize],
                                    CountType* smem,
                                    BitDataType desired,
                                    BitDataType desiredMask,
                                    int radixDigitPos,
                                    IndexType sliceSize,
                                    IndexType withinSliceStride,
                                    DataType* data) {
  // Clear out per-thread counts from a previous round
#pragma unroll
  for (int i = 0; i < RadixSize; ++i) {
    counts[i] = 0;
  }

  if (threadIdx.x < RadixSize) {
    smem[threadIdx.x] = 0;
  }
  __syncthreads();

  // Scan over all the data. Upon a read, the warp will accumulate
  // counts per each digit in the radix using warp voting.
  for (IndexType i = threadIdx.x; i < sliceSize; i += blockDim.x) {
    BitDataType val = conv.convert(doLdg(&data[i * withinSliceStride]));

    bool hasVal = ((val & desiredMask) == desired);
    unsigned int digitInRadix = getBitfield(val, radixDigitPos, RadixBits);

#pragma unroll
    for (unsigned int j = 0; j < RadixSize; ++j) {
      bool vote = hasVal && (digitInRadix == j);
      counts[j] += __popc(__ballot(vote));
    }
  }

  // Now, for each warp, sum values
  if (getLaneId() == 0) {
#pragma unroll
    for (unsigned int i = 0; i < RadixSize; ++i) {
      atomicAdd(&smem[i], counts[i]);
    }
  }

  __syncthreads();

  // For each thread, read in the total counts
#pragma unroll
  for (unsigned int i = 0; i < RadixSize; ++i) {
    counts[i] = smem[i];
  }

  __syncthreads();
}

// Over what radix we are selecting values
#define RADIX_BITS 2 // digits are base-(2 ^ RADIX_BITS)
#define RADIX_SIZE 4 // 2 ^ RADIX_BITS
#define RADIX_MASK (RADIX_SIZE - 1)

// This finds the unique value `v` that matches the pattern
// ((v & desired) == desiredMask) in our sorted int format
template <typename DataType, typename IndexType, typename RadixConverter>
__device__ float findPattern(const RadixConverter& conv,
                             DataType* smem,
                             DataType* data,
                             IndexType sliceSize,
                             IndexType withinSliceStride,
                             unsigned int desired,
                             unsigned int desiredMask) {
  if (threadIdx.x < 32) {
    smem[threadIdx.x] = (DataType) 0;
  }
  __syncthreads();

  // All threads participate in the loop, in order to sync on the flag
  IndexType numIterations = THCRoundUp(sliceSize, (IndexType) blockDim.x);
  for (IndexType i = threadIdx.x; i < numIterations; i += blockDim.x) {
    bool inRange = (i < sliceSize);
    DataType v = inRange ? doLdg(&data[i * withinSliceStride]) : (DataType) 0;

    if (inRange && ((conv.convert(v) & desiredMask) == desired)) {
      // There should not be conflicts if we are using findPattern,
      // since the result is unique
      smem[0] = (DataType) 1;
      smem[1] = v; // can't use val as the flag, since it could be 0
    }

    __syncthreads();

    DataType found = smem[0];
    DataType val = smem[1];

    __syncthreads();

    // Check to see if a thread found the value
    if (found != (DataType) 0) {
      // all threads return this value
      return val;
    }
  }

  // should not get here
  assert(false);
  return (DataType) 0;
}

// Returns the top-Kth element found in the data using radix selection
template <typename DataType, typename BitDataType, typename IndexType,
          typename RadixConverter, bool Order>
__device__ void radixSelect(const RadixConverter& conv,
                            DataType* data,
                            IndexType k,
                            IndexType sliceSize,
                            IndexType withinSliceStride,
                            int* smem,
                            DataType* topK) {
  // Per-thread buckets into which we accumulate digit counts in our
  // radix
  int counts[RADIX_SIZE];

  // We only consider elements x such that (x & desiredMask) == desired
  // Initially, we consider all elements of the array, so the above
  // statement is true regardless of input.
  unsigned int desired = 0;
  unsigned int desiredMask = 0;

  // We are looking for the top kToFind-th element when iterating over
  // digits; this count gets reduced by elimination when counting
  // successive digits
  int kToFind = k;

  // We start at the most significant digit in our radix, scanning
  // through to the least significant digit
#pragma unroll
  for (int digitPos = sizeof(BitDataType) * 8 - RADIX_BITS;
       digitPos >= 0;
       digitPos -= RADIX_BITS) {

    // Count radix distribution for the current position and reduce
    // across all threads
    countRadixUsingMask<DataType, BitDataType,
                        IndexType, int, RadixConverter,
                        RADIX_SIZE, RADIX_BITS>(
                          conv, counts, smem,
                          desired, desiredMask, digitPos,
                          sliceSize, withinSliceStride, data);

    // All threads participate in the comparisons below to know the
    // final result

#define CHECK_RADIX(i)                                                  \
    int count = counts[i];                                              \
                                                                        \
    /* All threads have the same value in counts here, so all */        \
    /* threads will return from the function. */                        \
    if (count == 1 && kToFind == 1) {                                   \
      /* There is a unique answer. */                                   \
      desired = setBitfield(desired, i, digitPos, RADIX_BITS);          \
      desiredMask =                                                     \
        setBitfield(desiredMask, RADIX_MASK, digitPos, RADIX_BITS);     \
                                                                        \
      /* The answer is now the unique element v such that: */           \
      /* (v & desiredMask) == desired */                                \
      /* However, we do not yet know what the actual element is. We */  \
      /* need to perform a search through the data to find the */       \
      /* element that matches this pattern. */                          \
      *topK = findPattern<DataType, IndexType, RadixConverter>(         \
        conv, (float*) smem, data, sliceSize,                           \
        withinSliceStride, desired, desiredMask);                       \
      return;                                                           \
    }                                                                   \
                                                                        \
    if (count >= kToFind) {                                             \
      desired = setBitfield(desired, i, digitPos, RADIX_BITS);          \
      desiredMask =                                                     \
        setBitfield(desiredMask, RADIX_MASK, digitPos, RADIX_BITS);     \
                                                                        \
      /* The top-Kth element v must now be one such that: */            \
      /* (v & desiredMask == desired) */                                \
      /* but we haven't narrowed it down; we must check the next */     \
      /* least-significant digit */                                     \
      break;                                                            \
    }                                                                   \
                                                                        \
    kToFind -= count                                                    \

    if (Order) {
      // Process in descending order
#pragma unroll
      for (int i = RADIX_SIZE - 1; i >= 0; --i) {
        CHECK_RADIX(i);
      }
    } else {
      // Process in ascending order
#pragma unroll
      for (int i = 0; i < RADIX_SIZE; ++i) {
        CHECK_RADIX(i);
      }
    }
#undef CHECK_RADIX
  } // end digitPos for

  // There is no unique result, but there is a non-unique result
  // matching `desired` exactly
  *topK = conv.deconvert(desired);
}

template <typename IndexType, int Dim, bool Order>
__global__ void gatherTopK(TensorInfo<float, IndexType> input,
                           IndexType inputSliceSize,
                           IndexType outputSliceSize, // aka `k`

                           IndexType numInputSlices,
                           IndexType inputWithinSliceStride,

                           TensorInfo<float, IndexType> topK,
                           IndexType numTopKSlices,
                           IndexType topKWithinSliceStride,

                           TensorInfo<long, IndexType> indices,
                           IndexType indicesWithinSliceStride) {
  // Indices are limited to integer fp precision, so counts can fit in
  // int32, regardless of IndexType
  __shared__ int smem[32]; // one per each warp, up to warp limit

  IndexType slice = getLinearBlockId<IndexType>();
  if (slice >= numInputSlices) {
    return;
  }

  // Find the start offset for our slice
  IndexType sliceStartIndex =
    IndexToOffset<float, IndexType, Dim>::get(slice, input);
  IndexType topKSliceStartIndex =
    IndexToOffset<float, IndexType, Dim>::get(slice, topK);
  IndexType indicesSliceStartIndex =
    IndexToOffset<long, IndexType, Dim>::get(slice, indices);

  float* inputSliceStart = &input.data[sliceStartIndex];
  float* topKSliceStart = &topK.data[topKSliceStartIndex];
  long* indicesSliceStart = &indices.data[indicesSliceStartIndex];

  // Find the k-th highest element in our input
  float topKValue = -1.0f;
  radixSelect<float, unsigned int, IndexType, FloatToSortedInt, Order>(
    FloatToSortedInt(),
    inputSliceStart, outputSliceSize,
    inputSliceSize, inputWithinSliceStride,
    smem, &topKValue);

  // Every value that is strictly less/greater than `pattern`
  // (depending on sort dir) in sorted int format is in the top-K.
  // The top-K value itself might not be unique.
  //
  // Since there are a variable number of elements that we see that
  // are within the top-k, we don't know at what index to write out
  // the resulting values.
  // In order to get this, we perform an exclusive prefix sum of
  // `hasTopK`. This will return the resulting index into which we
  // need to write the result, if a thread has a result.

  // All threads need to participate in the loop and the prefix sum,
  // but not necessarily in the load; hence loop bounds being rounded
  // up to a multiple of the block dim.
  IndexType numIterations = THCRoundUp(inputSliceSize, (IndexType) blockDim.x);
  IndexType writeIndexStart = 0;

  for (IndexType i = threadIdx.x; i < numIterations; i += blockDim.x) {
    bool inRange = (i < inputSliceSize);
    float v =
      inRange ? doLdg(&inputSliceStart[i * inputWithinSliceStride]) : 0.0f;
    bool hasTopK;
    if (Order) {
      hasTopK = inRange && (v > topKValue);
    } else {
      hasTopK = inRange && (v < topKValue);
    }

    int index;
    int carry;
    exclusiveBinaryPrefixSum<int, true>(smem, hasTopK, &index, &carry);

    if (hasTopK) {
      int writeIndex = writeIndexStart + index;
      assert(writeIndex < outputSliceSize);

      IndexType topKOffset = writeIndex * topKWithinSliceStride;
      IndexType indexOffset = writeIndex * indicesWithinSliceStride;

      topKSliceStart[topKOffset] = v;
      indicesSliceStart[indexOffset] = i + TH_INDEX_BASE; // to Lua index
    }

    writeIndexStart += carry;
  }

  // We need to fill in the rest with actual == top-K values.
  // The number that we need is outputSliceSize -
  // writeIndexStart. There might be more than that number available,
  // in which case we have to choose the first seen set. We do this
  // via a prefix sum to calculate indices for writing results.
  assert(outputSliceSize >= writeIndexStart);
  IndexType topKRemaining = (outputSliceSize - writeIndexStart);

  for (IndexType i = threadIdx.x; i < numIterations; i += blockDim.x) {
    bool inRange = (i < inputSliceSize);
    float v =
      inRange ? doLdg(&inputSliceStart[i * inputWithinSliceStride]) : 0.0f;
    bool hasTopK = inRange && (v == topKValue);

    int index;
    int carry;
    exclusiveBinaryPrefixSum<int, true>(smem, hasTopK, &index, &carry);

    if (hasTopK && index < topKRemaining) {
      int writeIndex = writeIndexStart + index;
      assert(writeIndex < outputSliceSize);

      IndexType topKOffset = writeIndex * topKWithinSliceStride;
      IndexType indexOffset = writeIndex * indicesWithinSliceStride;

      topKSliceStart[topKOffset] = v;
      indicesSliceStart[indexOffset] = i + TH_INDEX_BASE; // to Lua index
    }

    if (carry >= topKRemaining) {
      break;
    }

    topKRemaining -= carry;
    writeIndexStart += carry;
  }
}

#undef RADIX_BITS
#undef RADIX_SIZE
#undef RADIX_MASK

THC_API void THCudaTensor_topk(THCState* state,
                               THCudaTensor *topK,
                               THCudaLongTensor *indices,
                               THCudaTensor *input,
                               long k, int dim, int dir, int sorted) {
  THAssert(topK != NULL && indices != NULL && input != NULL);
  THAssert(THCudaTensor_checkGPU(state, 3, topK, indices, input));
  THCCheckTensorDims(state, topK, 2);
  long dims = THCudaLongTensor_nDimension(state, indices);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  THCCheckTensorDims(state, input, 2);

  int numDims = THCudaTensor_nDimension(state, input);
  THArgCheck(dim >= 0 && dim < numDims, 3, "dim not in range");

  long sliceSize = THCudaTensor_size(state, input, dim);
  THArgCheck(k > 0 && k <= sliceSize, 2, "k not in range for dimension");

  // Build the output size, which is the dim being selected set to
  // size k
  THLongStorage* topKSize = THCudaTensor_newSizeOf(state, input);
  THLongStorage_set(topKSize, dim, k);
  THCudaTensor_resize(state, topK, topKSize, NULL);
  THCudaLongTensor_resize(state, indices, topKSize, NULL);
  THLongStorage_free(topKSize);

#define RUN_K(INDEX_T, DIM, DIR)                                        \
  gatherTopK<INDEX_T, DIM, DIR>                                         \
    <<<grid, block, 0, THCState_getCurrentStream(state)>>>(             \
      inputInfo,                                                        \
      sliceSize,                                                        \
      k,                                                                \
      inputSlices,                                                      \
      /* The actual dimension that the k-selection is running in */     \
      /* may have changed from collapseDims() */                        \
      inputInfo.strides[collapseInputDim],                              \
      topKInfo,                                                         \
      topKSlices,                                                       \
      topKInfo.strides[collapseTopKDim],                                \
      indicesInfo,                                                      \
      indicesInfo.strides[collapseIndicesDim])

#define RUN_DIR(INDEX_T, DIM)                   \
  if (dir) {                                    \
    RUN_K(INDEX_T, DIM, true);                  \
  } else {                                      \
    RUN_K(INDEX_T, DIM, false);                 \
  }

#define RUN_DIM(INDEX_T)                        \
  if (allDims == 1) {                           \
    RUN_DIR(INDEX_T, 1);                        \
  } else if (allDims == 2) {                    \
    RUN_DIR(INDEX_T, 2);                        \
  } else if (allDims == 3) {                    \
    RUN_DIR(INDEX_T, 3);                        \
  } else {                                      \
    RUN_DIR(INDEX_T, -1);                       \
  }

#define RUN_T(INDEX_T)                                                  \
  TensorInfo<float, INDEX_T> inputInfo =                                \
    getTensorInfo<THCudaTensor, INDEX_T>(state, input);                 \
  TensorInfo<float, INDEX_T> topKInfo =                                 \
    getTensorInfo<THCudaTensor, INDEX_T>(state, topK);                  \
  TensorInfo<long, INDEX_T> indicesInfo =                               \
    getTensorInfo<THCudaLongTensor, INDEX_T>(state, indices);           \
                                                                        \
  /* We use these structures solely to find the offset to */            \
  /* each slice we are operating on */                                  \
  inputInfo.sizes[dim] = 1;                                             \
  topKInfo.sizes[dim] = 1;                                              \
  indicesInfo.sizes[dim] = 1;                                           \
                                                                        \
  /* Collapse all other dims */                                         \
  int collapseInputDim = inputInfo.collapseDims(dim);                   \
  int collapseTopKDim = topKInfo.collapseDims(dim);                     \
  int collapseIndicesDim = indicesInfo.collapseDims(dim);               \
                                                                        \
  long inputSlices = 1;                                                 \
  long topKSlices = 1;                                                  \
  for (int i = 0; i < numDims; ++i) {                                   \
    inputSlices *= inputInfo.sizes[i];                                  \
    topKSlices *= topKInfo.sizes[i];                                    \
  }                                                                     \
                                                                        \
  dim3 grid;                                                            \
  if (!THC_getGridFromTiles(inputSlices, grid)) {                       \
    THError("Slice to sort is too large");                              \
  }                                                                     \
                                                                        \
  dim3 block(std::min(THCRoundUp(sliceSize, 32L), 1024L));              \
                                                                        \
  /* This is used as a template parameter to calculate indices. */      \
  /* We only specialize it if all collapsed dim sizes are the */        \
  /* same; otherwise, we use -1 which is the specialization */          \
  /* parameter for arbitrary dimensions */                              \
  int allDims = inputInfo.dims;                                         \
  if (topKInfo.dims != allDims || indicesInfo.dims != allDims) {        \
    allDims = -1;                                                       \
  }                                                                     \
                                                                        \
  RUN_DIM(INDEX_T);

  // Based on required index size, run the algorithm with the
  // appropriate index type
  if (TensorUtils<THCudaTensor>::canUse32BitIndexMath(state, input) &&
      TensorUtils<THCudaTensor>::canUse32BitIndexMath(state, topK) &&
      TensorUtils<THCudaLongTensor>::canUse32BitIndexMath(state, indices)) {
    RUN_T(unsigned int);
  } else {
    RUN_T(unsigned long);
  }
#undef RUN_T
#undef RUN_DIM
#undef RUN_DIR
#undef RUN_K

  // Sort the results if the user wants them sorted, since our
  // selection routine does not ensure sorting
  if (sorted) {
    // FIXME: the k/v inplace sort along slice only works for size <=
    // 2048 at the moment
    if (sliceSize <= 2048) {
      // This avoids any memory allocations and performs all sorting
      // work inplace along the slice
      THCudaTensor_sortKeyValueInplace(state, topK, indices, dim, dir);
    } else {
      // Depend upon the backup sort that returns indices, which we
      // can use in conjunction with gather to produce the original
      // indices.
      // This is not the most efficient implementation, especially since
      // there are memory allocations performed here. If the user desires
      // greater performance, they should torch.gather() the results
      // themselves using the reported indices, providing previously
      // allocated tensors to receive the results.
      THCudaTensor* sortedTopK = THCudaTensor_new(state);
      THCudaLongTensor* sortedIndices = THCudaLongTensor_new(state);
      THCudaTensor_sort(state, sortedTopK, sortedIndices, topK, dim, dir);

      THCudaLongTensor* sortedTopKIndices = THCudaLongTensor_new(state);

      THCudaLongTensor_resizeAs(state, sortedTopKIndices, indices);
      THCudaLongTensor_gather(state, sortedTopKIndices, indices, dim, sortedIndices);

      THCudaTensor_freeCopyTo(state, sortedTopK, topK);
      THCudaLongTensor_freeCopyTo(state, sortedTopKIndices, indices);
      THCudaLongTensor_free(state, sortedIndices);
    }
  }

  THCudaCheck(hipGetLastError());
}
